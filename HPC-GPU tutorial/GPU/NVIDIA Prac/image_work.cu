#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "read_bmp.h"
#include "read_bmp_clib.h"
__global__ void kernel(uint8_t* in_img,uint8_t* out_img,int ht,int wd)
{
    int inX = blockDim.x*blockIdx.x+threadIdx.x;
    int inY = blockDim.y*blockIdx.y+threadIdx.y;
    int outX = wd-inY-1;
    int outY = inX;
    out_img[outX*wd+outY] = in_img [inX*wd+inY];
}
int main(int argc, char** argv)
{
    //1. read the file
	char* input_img_file = argv[1];
    char* cuda_img = argv[2];
    
    //2. allocate host memory
    
	//============== reading the binary bmp file into buffer ==============
	FILE *input_file;
    input_file = fopen(input_img_file,"rb");
    bmp_image img;
	uint8_t *h_bmp_data;
    h_bmp_data = (uint8_t *)img.read_bmp_file(input_file);
    //allocate host memory for result
    uint8_t *h_bmp_result;
    h_bmp_result = (uint8_t*)malloc(img.num_pixel);
    //get image attributes
    int wd = img.image_width;	int ht = img.image_height;
    //3. thread hierarchy: grid is 32*32 blocks. Each block is 32*32 threads. Each thread handles one pixel
        int gridDimX = wd/32;
        int gridDimY = ht/32;
        int blockDimX = 32;
        int blockDimY = 32;

    //4. allocate the Device memory 
    uint8_t *d_bmp_data;
    uint8_t *d_bmp_result;
    hipMalloc((void**)&d_bmp_data, img.num_pixel);
    hipMalloc((void**)&d_bmp_result, img.num_pixel);

    //5. pass the data to GPU
    hipMemcpy(d_bmp_data,h_bmp_data,img.num_pixel,hipMemcpyHostToDevice);

    //6. run the kernel
    dim3 dimBlock(blockDimX, blockDimY);
    dim3 dimGrid(gridDimX,gridDimY);
    kernel<<<dimBlock, dimGrid>>>(d_bmp_data,d_bmp_result,ht,wd);

    //7. get the result back to the host
    hipMemcpy(h_bmp_result,d_bmp_result,img.num_pixel,hipMemcpyDeviceToHost);

	//8. free GPU memory
	hipFree( h_bmp_data );
	hipFree( d_bmp_result);

	//9. write back the new bmp image into output file
	FILE *output_file1;
	output_file1 = fopen(cuda_img,"wb");
    img.write_bmp_file(output_file1, h_bmp_result);

    return 0;
}