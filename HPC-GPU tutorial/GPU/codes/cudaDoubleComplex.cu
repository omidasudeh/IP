// L-30 MCS 572 Mon 31 Oct 2016 : cudaDoubleComplex.cu
// Our complex double precision numbers use the double2 type,
// double2 is a native CUDA type allowing for coalesced 256 byte access.

#ifndef __CUDADOUBLECOMPLEX_CU__
#define __CUDADOUBLECOMPLEX_CU__

#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <hip/hip_vector_types.h>
#include <math_functions.h>

typedef double2 cudaDoubleComplex;

__host__ cudaDoubleComplex randomDoubleComplex()
// Returns a complex number on the unit circle
// with angle uniformly generated in [0,2*pi].
{
   cudaDoubleComplex result;
   int r = rand();
   double u = double(r)/RAND_MAX;
   double angle = 2.0*M_PI*u;
   result.x = cos(angle);
   result.y = sin(angle);
   return result;
}

__device__ double radius ( const cudaDoubleComplex c )
// Returns the radius of the complex number.
{
   double result;
   result = c.x*c.x + c.y*c.y;
   return sqrt(result);
}

__host__ std::ostream& operator<<
 ( std::ostream& os, const cudaDoubleComplex& c)
// Writes real and imaginary parts of c,
// in scientific notation with precision 16.
{
   os << std::scientific << std::setprecision(16)
      << c.x << "  " << c.y;
   return os;
}

__device__ cudaDoubleComplex operator+
 ( const cudaDoubleComplex a, const cudaDoubleComplex b )
// Returns the sum of a and b.
{
   cudaDoubleComplex result;
   result.x = a.x + b.x;
   result.y = a.y + b.y;
   return result;
}

__device__ cudaDoubleComplex operator-
 ( const cudaDoubleComplex a, const cudaDoubleComplex b )
// Returns the difference of a minus b.
{
   cudaDoubleComplex result;
   result.x = a.x - b.x;
   result.y = a.y - b.y;
   return result;
}

__device__ cudaDoubleComplex operator* 
 ( const cudaDoubleComplex a, const cudaDoubleComplex b )
// Returns the product of a and b.
{
   cudaDoubleComplex result;
   result.x = a.x*b.x - a.y*b.y;
   result.y = a.x*b.y + a.y*b.x;
   return result;
}

__device__ cudaDoubleComplex operator/
 ( const cudaDoubleComplex a, const cudaDoubleComplex b )
// Returns the division of a by b.
{
   cudaDoubleComplex result;
   double d = b.x*b.x + b.y*b.y;
   result.x = (a.x*b.x + a.y*b.y)/d;
   result.y = (a.y*b.x - a.x*b.y)/d;
   return result;
}

#endif
