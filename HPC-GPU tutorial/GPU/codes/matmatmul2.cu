#include "hip/hip_runtime.h"
/* L-31 MCS 572 Wed 2 Nov 2016 : matmatmul2.cu 
 * This file contains a very basic CUDA implementation 
 * of the multiplication of two 0/1 matrices.
 * At the command line the user must give three dimensions: n, m, and p.
 * The program generates a random n-by-m 0/1 matrix A,
 * a random m-by-p 0/1 matrix B, and then computes the product.
 * This version uses the x and y component of threadIdx.  */

#include <stdio.h>
#include <stdlib.h>

__host__ void randomMatrix ( int n, int m, float *x )
/* 
 * Fills up the n-by-m matrix x with random
 * values of zeroes and ones. */
{
   int i,j,r;
   float *p = x;
   for(i=0; i<n; i++)
      for(j=0; j<m; j++)
      {
         r = rand() % 2;
         *(p++) = (float) r;
      }
}

__host__ void writeMatrix ( int n, int m, float *x )
/*
 * Writes the n-by-m matrix x to screen. */
{
   int i,j;
   float *p = x;
   for(i=0; i<n; i++,printf("\n"))
      for(j=0; j<m; j++)
         printf(" %d", (int)*(p++));
}

__global__ void matrixMultiply
 ( int n, int m, int p, float *A, float *B, float *C )
/*
 * Multiplies the n-by-m matrix A 
 * with the m-by-p matrix B into the matrix C.
 * The (i,j)-th thread computes the (i,j)-th element of C. */
{
   int i = threadIdx.x;
   int j = threadIdx.y;
   int ell = i*p + j;
   C[ell] = 0.0;
   float *pB;
   for(int k=0; k<m; k++)
   {
      pB = &B[j+k*p];
      C[ell] += A[i*m+k]*(*pB);
   }
}

int main ( int argc, char*argv[] )
{
   if(argc < 4)
   {
      printf("call with 3 arguments :\n");
      printf("dimensions n, m, and p\n");
   }
   else
   {
      int n = atoi(argv[1]); /* number of rows of A */
      int m = atoi(argv[2]); /* number of columns of A */
                             /* and number of rows of B */
      int p = atoi(argv[3]); /* number of columns of B */
      srand(time(0));
      printf("a random %d-by-%d 0/1 matrix A :\n",n,m);
      float *Ahost = (float*)calloc(n*m,sizeof(float));
      randomMatrix(n,m,Ahost); writeMatrix(n,m,Ahost);
      printf("a random %d-by-%d 0/1 matrix B :\n",m,p);
      float *Bhost = (float*)calloc(m*p,sizeof(float));
      randomMatrix(m,p,Bhost); writeMatrix(m,p,Bhost);
      float *Chost = (float*)calloc(n*p,sizeof(float));
      /* allocate memory on the device for A, B, and C */
      float *Adevice;
      size_t sA = n*m*sizeof(float);
      hipMalloc((void**)&Adevice,sA);
      float *Bdevice;
      size_t sB = m*p*sizeof(float);
      hipMalloc((void**)&Bdevice,sB);
      float *Cdevice;
      size_t sC = n*p*sizeof(float);
      hipMalloc((void**)&Cdevice,sC);
      /* copy matrices A and B from host to the device */
      hipMemcpy(Adevice,Ahost,sA,hipMemcpyHostToDevice);
      hipMemcpy(Bdevice,Bhost,sB,hipMemcpyHostToDevice);
      /* kernel invocation launching n*p threads */
      dim3 dimGrid(1,1);
      dim3 dimBlock(n,p);
      matrixMultiply<<<dimGrid,dimBlock>>>(n,m,p,Adevice,Bdevice,Cdevice);
      /* copy matrix C from device to the host */
      hipMemcpy(Chost,Cdevice,sC,hipMemcpyDeviceToHost);
      /* freeing memory on the device */
      hipFree(Adevice); hipFree(Bdevice); hipFree(Cdevice);
      printf("the resulting %d-by-%d matrix C :\n",n,p);
      writeMatrix(n,p,Chost);
   }
   return 0;
}
