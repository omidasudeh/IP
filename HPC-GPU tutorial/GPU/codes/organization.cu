/* L-33 MCS 572 Mon 7 Nov 2016 : organization.cu 
 * This simple program illustrates a kernel launching a three dimensional
 * grid of three dimensional blocks. */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void matrixFill ( int *x )
/*
 * Fills the matrix using blockIdx and threadIdx. */
{
   int bx = blockIdx.x;
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;
   int row = by*blockDim.y + ty;
   int col = bx*blockDim.x + tx;
   int dim = gridDim.x*blockDim.x;
   int i = row*dim + col;
   x[i] = i; 
}

int main ( int argc, char* argv[] )
{
   const int xb = 2; /* gridDim.x */
   const int yb = 2; /* gridDim.y */
   const int zb = 1; /* gridDim.z */
   const int xt = 2; /* blockDim.x */
   const int yt = 2; /* blockDim.y */
   const int zt = 1; /* blockDim.z */
   const int n = xb*yb*zb*xt*yt*zt;
   printf("allocating array of length %d...\n",n);
   /* allocate, initialize, and copy to device */
   int *xhost = (int*)calloc(n,sizeof(int));
   for(int i=0; i<n; i++) xhost[i] = -1.0;
   int *xdevice;
   size_t sx = n*sizeof(int);
   hipMalloc((void**)&xdevice,sx);
   hipMemcpy(xdevice,xhost,sx,hipMemcpyHostToDevice);
   /* set the execution configuration for the kernel */
   dim3 dimGrid(xb,yb,zb);
   dim3 dimBlock(xt,yt,zt);
   matrixFill<<<dimGrid,dimBlock>>>(xdevice);
   /* copy data from device to host */
   hipMemcpy(xhost,xdevice,sx,hipMemcpyDeviceToHost);
   hipFree(xdevice);
   int *p = xhost;
   for(int i1=0; i1 < xb; i1++)
      for(int i2=0; i2 < yb; i2++)
         for(int i3=0; i3 < zb; i3++)
            for(int i4=0; i4 < xt; i4++)
               for(int i5=0; i5 < yt; i5++)
                  for(int i6=0; i6 < zt; i6++)
                     printf("x[%d][%d][%d][%d][%d][%d] = %d\n",
                             i1,i2,i3,i4,i5,i6,*(p++));
   return 0;
}
