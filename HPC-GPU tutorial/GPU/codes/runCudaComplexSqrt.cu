#include "hip/hip_runtime.h"
// L-30 MCS 572 Fri 31 Oct 2016 : runCudaComplexSqrt.cu
// Using a GPU to compute the square root of n random complex numbers.

#include <cstdlib>
#include <ctime>
#include <iostream>
#include <iomanip>
#include "cudaDoubleComplex.cu"

using namespace std;

__global__ void squareRoot
 ( int n, cudaDoubleComplex *x, cudaDoubleComplex *y )
// Applies Newton's method to compute the square root 
// of the n numbers in x and places the results in y.
{
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   cudaDoubleComplex inc;
   cudaDoubleComplex c = x[i];
   cudaDoubleComplex r = c;
   for(int j=0; j<5; j++)
   {
      inc = r + r;
      inc = (r*r - c)/inc;
      r = r - inc;
   }
   y[i] = r;
}

cudaDoubleComplex Square ( const cudaDoubleComplex z )
// Returns z*z for testing purposes.
{
   cudaDoubleComplex result;
   result.x = z.x*z.x - z.y*z.y;
   result.y = 2*z.x*z.y;
   return result;
}

int main ( int argc, char*argv[] )
{
   if(argc < 5)
   {
      cout << "call with 4 arguments : " << endl;
      cout << "dimension, block size, frequency, and check (0 or 1)" << endl;
   }
   else
   {
      int n = atoi(argv[1]); // dimension
      int w = atoi(argv[2]); // block size
      int f = atoi(argv[3]); // frequency
      int t = atoi(argv[4]); // test or not
      // we generate n random complex numbers on the host
      cudaDoubleComplex *xhost = new cudaDoubleComplex[n];
      for(int i=0; i<n; i++) xhost[i] = randomDoubleComplex();
      // we copy the n random complex numbers to the device
      size_t s = n*sizeof(cudaDoubleComplex);
      cudaDoubleComplex *xdevice;
      hipMalloc((void**)&xdevice,s);
      hipMemcpy(xdevice,xhost,s,hipMemcpyHostToDevice);
      // allocate memory for the result
      cudaDoubleComplex *ydevice;
      hipMalloc((void**)&ydevice,s);
      // invoke the kernel with n/w blocks per grid
      // and w threads per block
      for(int i=0; i<f; i++)
         squareRoot<<<n/w,w>>>(n,xdevice,ydevice);
      // copy results from device to host
      cudaDoubleComplex *yhost = new cudaDoubleComplex[n];
      hipMemcpy(yhost,ydevice,s,hipMemcpyDeviceToHost);
      if(t == 1) // test the result
      {
         int k = rand() % n;
         cout << "testing number " <<  k << endl;
         cout << "        x = " << xhost[k] << endl;
         cout << "  sqrt(x) = " << yhost[k] << endl;
         cudaDoubleComplex z = Square(yhost[k]);
         cout << "sqrt(x)^2 = " << z << endl;
      }
   }
   return 0;
}
