#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
__global__ void MMM(float* A, float*B, float*C, int m, int w)
{
//    extern __shared__ int a[];
    __shared__ float a_tile[32][32];
    __shared__ float b_tile[32][32];// should I use padding to avoid shared memory bank conflicts
    int by = blockIdx.y;          int bx = blockIdx.x;
    int ty = by*w+threadIdx.y;    int tx =bx*w+threadIdx.x;
    int cval = 0;
    for(int k = 0; k<=m/w;k++)
    {
        //each thread load its element of A and B into shared memory tiles
        // load tile a_tile = A[by][k], b_tile = B[k][bx]
        a_tile[threadIdx.y][threadIdx.x] = A[(by*w+threadIdx.y)*m // same for all threads in the warp
                                             +k*w+threadIdx.x];// access to conseq elelments in global memory
        b_tile[threadIdx.y][threadIdx.x] = B[(k*w+threadIdx.y)*m
                                             +bx*w+threadIdx.x];
        __syncthreads();//why?   
        for(int i = 0;i<w;i++)
            cval+= a_tile[threadIdx.y][i]*b_tile[i][threadIdx.x];
        __syncthreads();//why? 
    }
    C[ty*m+tx] = cval;       
}
void print(float* a, int dim)
{
    printf("\n");
    for (int i = 0;i<dim;i++){
        for (int j = 0;j<dim;j++)
            printf("%f   ",a[i*dim+j]);
        printf("\n");
    }

}
int main(int argc, char** argv)
{
    //1. allocate and initiate host data
        float *H_A, *H_B, *H_C;
        int dim = 1024*4;
        int m = dim*dim;
        int memory_size = m*sizeof(float);
        H_A = (float*) malloc(memory_size);
        H_B = (float*) malloc(memory_size);
        H_C = (float*) malloc(memory_size);
        time_t t;
        srand((unsigned) time(&t));
        for(int i = 0;i<m;i++)
        {
            float ta  = rand()%5;
            H_A[i] = ta;
            float tb =  rand()%5;
            H_B[i] = tb;
        }
    //2. allocate and initiate device data
        float *D_A, *D_B, *D_C;
        hipMalloc((void**)&D_A, memory_size);
        hipMalloc((void**)&D_B, memory_size);
        hipMalloc((void**)&D_C, memory_size);
    //3. thread hierarchy
        int by, bx, gy, gx;
        int bdim = 32;
        int gdim = dim/bdim;
        by = bdim; bx = bdim;
        gy = gdim; gx = gdim;
        dim3 dimBlock(by, bx);
        dim3 dimGrid(gy,gx);
    //4. transfer data from host to device
        hipMemcpy(D_A,H_A,memory_size,hipMemcpyHostToDevice);
        hipMemcpy(D_B,H_B,memory_size,hipMemcpyHostToDevice);
    //5. run the kernel
        MMM<<<dimGrid,dimBlock>>>(D_A,D_B,D_C,dim, bdim);
    //6. transfer result from device to host
        hipMemcpy(H_C,D_C,memory_size,hipMemcpyDeviceToHost);        
    //7. show the result
        // print(H_C);
        // print(H_A, dim);
        // print(H_B, dim);
        // print(H_C, dim);
    //8. free the device memory
        hipFree(D_A);
        hipFree(D_B);
        hipFree(D_C);

    return 0;
}