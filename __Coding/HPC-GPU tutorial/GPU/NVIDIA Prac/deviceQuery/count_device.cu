/*
*
*
*
This program illustrates counting available devices,
compile it with nvcc and note the extension .cu
and for a more detailed version, see deviceQuery.cpp
of the GPU Computing SDK */

#include <hip/hip_runtime.h>
#include <stdio.h>
void printDeviceProp ( hipDeviceProp_t p )
/*
* prints some device properties */
{
    printf(" name : %s \n",p.name);
    printf(" number of multiprocessors : %d \n",
    p.multiProcessorCount);
}
int main ( int argc, char* argv[] )
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("number of devices : %d\n",deviceCount);
    for(int d = 0; d < deviceCount; d++)
    {
        hipDeviceProp_t dev;
        hipGetDeviceProperties(&dev,d);
        printf("graphics card %d :\n",d);
        printDeviceProp(dev);
    }
    return 0;
}